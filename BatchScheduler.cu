//do some include stuff
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "matrixMul_kernel.cu"
#include "sleep_kernel.cu"

// set the default value of the kernel time to 1 second


/////////////////////////////////////////////////////////////////
// Global Variables
/////////////////////////////////////////////////////////////////

int kernel_time = 1000;

struct timeval tp;

double startTime_ms;


////////////////////////////////////////////////////////////////
// Utilities
////////////////////////////////////////////////////////////////

double getTime_msec() {
   gettimeofday(&tp, NULL);
   return static_cast<double>(tp.tv_sec) * 1E3
           + static_cast<double>(tp.tv_usec) / 1E3;
}

char* getNextKernel()
{
    return "sleep";
}

void call(char *kernel, hipStream_t stream)
{
    if(kernel=="sleep")
    {
        sleep(stream, kernel_time);
    }
}

void printAnyErrors()
{
    hipError_t e = hipGetLastError();
    if(e!=hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString( e ) );
    }
}



////////////////////////////////////////////////////////////////////
// The Main
////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
    startTime_ms = getTime_msec();

    int throttle = 16;  //this should be set using device properties

    int jobs = 64;

    if( argc>3 ){
        throttle = atoi(argv[1]);
        jobs = atoi(argv[2]);
        kernel_time = atoi(argv[3]);
    }    

    hipStream_t *streams = (hipStream_t *) malloc(throttle*sizeof(hipStream_t));

    for(int i = 0; i < throttle; i++)
    {
      hipStreamCreate(&streams[i]);
    }

    char *kernel = "none";

    printf("starting\n");

    printf("The number of jobs equals: %d\n",jobs);
    printf("The current throttle is: %d\n", throttle);
    int est = (((jobs-1)/throttle)+1)*kernel_time;
    printf("The estimated time should be: %d\n\n",est);

    for(int k = 0; k<jobs;) //later will probably just be true.
    {
        for(int j=0;j<throttle && k<jobs;j++){
            while( kernel == "none" ){
                kernel = getNextKernel();
            }
            call(kernel, streams[j]);
            k++;

            kernel = "none";
        }
        hipDeviceSynchronize();
    }

    // release resources

    for(int i =0; i<throttle; i++) hipStreamDestroy(streams[i]);

    free(streams);

    return 0;    
}






