
#include <hip/hip_runtime.h>
//declar globals and structs
struct timeval tp;
double getTime_sec();
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, float);
bool check(float*, float*, int, float);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

//    END OF KERNEL                                                                                                                                                                                        
// start CUDA mat mul

template <int BLOCK_SIZE> __global__ void
cudaMatrixMul( float* C, float* A, float* B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}
 // Allocates a matrix with random float entries.                                                                                            


void matrixMul(hipStream_t s, int side_length, float* d_arrayA, float* d_result){

  printf("IN matrixMul\n");
  int size = side_length;

  int block_size = 32;

  // setup execution parameters                                                           
  dim3 threads(block_size, block_size);
  dim3 grid(side_length / threads.x, side_length / threads.y);
                                                                                                                                                                        
  printf("Calculating: C = A x A where all sides equal: %d\n", side_length);

  // call the cudaMatrixMul cuda function
    cudaMatrixMul<32><<< grid, threads >>>(d_result, d_arrayA, d_arrayA, side_length, side_length);
    //int hard_coded = 100;
    //cudaMatrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, hard_coded, hard_coded);
}






