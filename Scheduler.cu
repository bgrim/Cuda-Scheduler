//do some include stuff
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "matrixMul_kernel.cu"
#include "sleep_kernel.cu"
#include "queue.c"
#include <pthread.h>

// set the default value of the kernel time to 1 second


/////////////////////////////////////////////////////////////////
// Global Variables
/////////////////////////////////////////////////////////////////

int kernel_time = 1000;

struct timeval tp;

double startTime_ms;

Queue Q;
pthread_mutex_t queueLock;


////////////////////////////////////////////////////////////////
// Utilities
////////////////////////////////////////////////////////////////

double getTime_msec() {
   gettimeofday(&tp, NULL);
   return static_cast<double>(tp.tv_sec) * 1E3
           + static_cast<double>(tp.tv_usec) / 1E3;
}

record* getStream()
{
    bool waiting = true;
    record *r;
    //hipStream_t stream;
    while(waiting)
    {
        pthread_mutex_lock(&queueLock);
        waiting = IsEmpty(Q);
        if(!waiting)
        {
	  //stream = Front(Q);
	    r = Front(Q);
            Dequeue(Q);
        }
        pthread_mutex_unlock(&queueLock);
        if(waiting) pthread_yield();
    }
    return r;
}

void putStream(record *r){
    //bool waiting = true;
    //while(waiting)
    //{
        pthread_mutex_lock(&queueLock);
        //waiting = IsFull(Q);
        //if(!waiting) Enqueue(stream, Q);
        Enqueue(r, Q);    //extra line
        pthread_mutex_unlock(&queueLock);
        //if(waiting) pthread_yield();
    //}
}

void *waitOnStream( void *arg )
{
  //    hipStream_t stream = (hipStream_t) arg;
    record *r = (record *) arg;

    // hipEvent_t event;
    // hipEventCreate(&event); 
    // hipEventRecord(event, stream);
    // hipEventSynchronize(event);
    // hipEventDestroy(event);
    
    //hipStreamSynchronize(r->stream);

    double time = getTime_msec();
    while(hipSuccess!=hipStreamQuery(r->stream)){
        //while(getTime_msec()<time+500);
    }

    printf(" done waiting for kernel at %.4f ms in stream: %d\n",getTime_msec() - startTime_ms, r->index);

    putStream(r);

    return 0;
}

char* getNextKernel()
{
    return "sleep";
}

void call(char *kernel)
{
    if(kernel=="sleep")
    {
        record *r = getStream();
        printf("   main at time %.2f ms in stream: %d\n", getTime_msec()-startTime_ms, r->index);
        pthread_t manager;
        sleep(r->stream, kernel_time);
        pthread_create( &manager, NULL, waitOnStream, (void *) r);
    }
}

void printAnyErrors()
{
    hipError_t e = hipGetLastError();
    if(e!=hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString( e ) );
    }
}



////////////////////////////////////////////////////////////////////
// The Main
////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
    startTime_ms = getTime_msec();

    pthread_mutex_init(&queueLock, NULL);

    int throttle = 16;  //this should be set using device properties

    int jobs = 64;

    Q = CreateQueue(throttle);

    if( argc>3 ){
        throttle = atoi(argv[1]);
        jobs = atoi(argv[2]);
        kernel_time = atoi(argv[3]);
    }    

    hipStream_t *streams = (hipStream_t *) malloc(throttle*sizeof(hipStream_t));

    // create record array
    record **recordArray = (record **) malloc(throttle*sizeof(record *));


    for(int i = 0; i < throttle; i++)
    {
      // create a new record with the cuda stream create and the loop counter as the index
      hipStreamCreate(&streams[i]);
      // allocate the record
      record *r = (record *) malloc (sizeof(struct record));
      r->stream = streams[i];
      r->index = i;
      Enqueue(r, Q);
      recordArray[i] = r;
    }

    char *kernel = "none";

    printf("starting\n");

    printf("The number of jobs equals: %d\n",jobs);
    printf("The current throttle is: %d\n", throttle);
    int est = (jobs/throttle)*kernel_time;
    printf("The estimated time should be: %d\n\n",est);

    for(int k = 0; k<jobs; k++) //later will probably just be true.
    {
        //while( kernel == "none" ){
        //    kernel = getNextKernel();
        //}
        kernel = "sleep";
        call(kernel);
        kernel = "none";
    }
                                                                                                                                                        
    

    hipError_t cuda_error = hipDeviceSynchronize();
    if(cuda_error==hipSuccess){
        //printf( "Final: Running the Scheduler was a success\n");
    }else{
        printf("Final: CUDA Error: %s\n", hipGetErrorString(cuda_error));
        return 1;
    }
    // release resources

    for(int i =0; i<throttle; i++) hipStreamDestroy(streams[i]);

    // free each element of the array
    for(int i = 0; i<throttle; i++)
    {
        free(recordArray[i]);
    }
    free(recordArray);

    free(streams);
    DisposeQueue(Q);
    
    for(int i=0; i<100000; i++);
    
    pthread_mutex_destroy(&queueLock);
    return 0;    
}






