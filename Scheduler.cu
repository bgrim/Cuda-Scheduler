//do some include stuff
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "matrixMul_kernel.cu"
#include "sleep_kernel.cu"
#include "queue.c"
#include <pthread.h>

// set the default value of the kernel time to 1 second


/////////////////////////////////////////////////////////////////
// Global Variables
/////////////////////////////////////////////////////////////////

int kernel_time = 1000;

double startTime_ms;
struct timeval tp;

Queue Q;
pthread_mutex_t queueLock;


////////////////////////////////////////////////////////////////
// Utilities
////////////////////////////////////////////////////////////////

double getTime_msec() {
   gettimeofday(&tp, NULL);
   return static_cast<double>(tp.tv_sec) * 1E3
           + static_cast<double>(tp.tv_usec) / 1E3;
}

hipStream_t getStream()
{
    bool waiting = true;
    hipStream_t stream;
    while(waiting)
    {
        pthread_mutex_lock(&queueLock);
        waiting = IsEmpty(Q);
        if(!waiting)
        {
            stream = Front(Q);
            Dequeue(Q);
        }
        pthread_mutex_unlock(&queueLock);
        if(waiting) pthread_yield();
    }
    return stream;
}

void putStream(hipStream_t stream){
    pthread_mutex_lock(&queueLock);
    Enqueue(stream, Q);
    pthread_mutex_unlock(&queueLock);
}

void *waitOnStream( void *str )
{
    hipStream_t stream = (hipStream_t) str;
    hipError_t cuda_error = hipStreamSynchronize(stream);
    
    putStream(stream);

    if(cuda_error==hipSuccess){
        printf( "A thread has finished in  %.4f ms\n", getTime_msec() - startTime_ms);
    }else{
        printf("CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }
    return 0;
}

char* getNextKernel()
{
    return "sleep";
}

void call(char *kernel)
{
    if(kernel=="sleep")
    {
        hipStream_t stream = getStream();
        sleep(stream, kernel_time);

        pthread_t thread1;
        int rc = pthread_create( &thread1, NULL, waitOnStream, (void *) stream);
    }
}

void printAnyErrors()
{
    hipError_t e = hipGetLastError();
    if(e!=hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString( e ) );
    }
}



////////////////////////////////////////////////////////////////////
// The Main
////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
    startTime_ms = getTime_msec();

    pthread_mutex_init(&queueLock, NULL);

    int throttle = 16;  //this should be set using device properties

    int jobs = 64;

    Q = CreateQueue(throttle);

    if( argc>3 ){
        throttle = atoi(argv[1]);
        jobs = atoi(argv[2]);
        kernel_time = atoi(argv[3]);
    }

    // allocate and initialize an array of stream handles
    //hipStream_t *streams = (hipStream_t*) malloc(throttle * sizeof(hipStream_t));
    
    for(int i = 0; i < throttle; i++)
    {
        hipStream_t s;
        hipStreamCreate(&s);
        Enqueue(s, Q);
    }

    char *kernel = "none";

    printf("starting\n");

    for(int k = 0; k<jobs; k++) //later will probably just be true.
    {
        while( kernel == "none" ){
            kernel = getNextKernel();
        }
        call(kernel);
        kernel = "none";
    }
    
    // print out some default information                                                                                                                                                                  
    printf("The number of jobs equals: %d\n",jobs);
    printf("The current throttle is: %d\n", throttle);
    int est = (jobs/throttle)*kernel_time;
    printf("The estimated time should be: %d\n",est);

    hipError_t cuda_error = hipDeviceSynchronize();
    if(cuda_error==hipSuccess){
        //printf( "Final: Running the Scheduler was a success\n");
    }else{
        printf("Final: CUDA Error: %s\n", hipGetErrorString(cuda_error));
        return 1;
    }
    // release resources
    DisposeQueue(Q);

    pthread_mutex_destroy(&queueLock);
    return 0;    
}






