#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "matrixMul_kernel.cu"
#include "sleep_kernel.cu"
#include "daemon.c"
#include <pthread.h>

/////////////////////////////////////////////////////////////////
// Global Variables
/////////////////////////////////////////////////////////////////

double startTime_ms;  //this is helpful for debugging sometimes
                      //its vlue is the first thing set by the program
struct tp;
////////////////////////////////////////////////////////////////
// Utilities
////////////////////////////////////////////////////////////////
double getTime_msec() {
   gettimeofday(&tp, NULL);
   return static_cast<double>(tp.tv_sec) * 1E3
           + static_cast<double>(tp.tv_usec) / 1E3;
}

//This method will let whatever kernel is about to run setup any device memory it needs
//  and do any file I/O needed. All Asynchronous operations will be in stream
void *kernel_setup(int kernel, hipStream_t stream, char * filename)
{
    if(kernel==1) return sleep_setup(stream, filename);

    if(kernel==2) return matMul_setup(stream, filename);

    return (void *) 1;
}

//This method will launch the given kernel in stream with setupResults.
void kernel_call(int kernel, hipStream_t stream, void *setupResults)
{
    if(kernel==1) sleep(stream, setupResults);

    if(kernel==2) matrixMul(stream, setupResults);
}

//This method will let the kernel deallocate all the memory that it acquired in
//  kernel_setup and also lets the kernel write to its output file.
void kernel_finish(int kernel, char * filename, void *setupResult )
{
    if(kernel==1) sleep_finish(filename, setupResult);

    if(kernel==2) matMul_finish(filename, setupResult);
}


//prints the most recent error that hasn't been printed before
//does nothing if there are no errors
void printAnyErrors()
{
    hipError_t e = hipGetLastError();
    if(e!=hipSuccess){
        printf("CUDA Error: %s\n", hipGetErrorString( e ) );
    }
}

////////////////////////////////////////////////////////////////////
// The Main
////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
    startTime_ms = getTime_msec();

    //sets the throttle and number of jobs based on inputs or defaults
    int throttle = 16;
    int jobs = 64;
    if( argc>2 ){
        throttle = atoi(argv[1]);
        jobs = atoi(argv[2]);
    }

    daemon_init(jobs);
    pthread_t daemon;
    pthread_create(&daemon, NULL, daemon_Main, (void *) &jobs );

    printf("The number of jobs is equal to: %d\n", jobs);

    //make throttle many streams to run concurrent kernels in
    hipStream_t *streams = (hipStream_t *) malloc(throttle*sizeof(hipStream_t));
    for(int i = 0; i < throttle; i++)
	hipStreamCreate(&streams[i]);

    int batchNum = 0;

    // loop for number of batches
    for(int k = 0; k<jobs;)
    {
        int batchSize = 0; //this will be throttle or less if we run out of jobs

	// arrays for kernel type and its input/output files
	int *kernels = (int *) malloc(throttle*sizeof(int));
	char **inputFiles = (char **) malloc(throttle*sizeof(char *));
	char **outputFiles = (char **) malloc(throttle*sizeof(char *));

        // get information for throttle many jobs or until we are out of jobs
	for(int q=0; q<throttle && k<jobs; q++){
	    kernels[q] = daemon_GetNextKernel();
	    inputFiles[q] = daemon_GetInputFile();
            outputFiles[q] = daemon_GetOutputFile();

	    printf("Kernel information for kernel  %d  of batch number  %d\n", q, batchNum);
	    printf("kernel: %d\n", kernels[q]);
	    printf("input:  %s\n", inputFiles[q]);
	    printf("output: %s\n\n", outputFiles[q]);

            k++;
            batchSize++;
	}

	// An array containing the state that each kernel needs
	void **setupResults = (void **) malloc(throttle*sizeof(void *));

	// Let each kernel read its input file and fill its setupResult
	for(int q=0; q<batchSize; q++){
	    setupResults[q] = kernel_setup(kernels[q], streams[q], inputFiles[q]);
	}

	// call each kernel in a different stream giving it its setupResult
        for(int q=0; q<batchSize; q++){
            kernel_call(kernels[q], streams[q], setupResults[q]);
        }

	// wait for all kernels to finish
        hipError_t err = hipDeviceSynchronize();
	printf("kernels finished kernels with error: %s\n", hipGetErrorString( err ) );

	// let each kernel copy its results back and write to its output file
	// they should do there own clean up (i.e. memory deallocate and closing files)
	for(int q=0; q<batchSize; q++){
	    kernel_finish(kernels[q], outputFiles[q], setupResults[q]);
	}


	//these values were allocated by the daemon but need to be deallocated
	for(int q=0;q<batchSize;q++){
	    free(inputFiles[q]);
	    free(outputFiles[q]);
	}

	//free the arrays that we used;
	free(kernels);
	free(inputFiles);
	free(outputFiles);

	printf("finished batch number: %d\n\n", batchNum);

        batchNum++;
    }

    hipError_t err = hipDeviceSynchronize();
    printf("finished all jobs with error: %s\n\n", hipGetErrorString( err ) );
    // release resources

    printf("The number of jobs equals: %d\n",jobs);
    printf("The current throttle is: %d\n", throttle);

    for(int i =0; i<throttle; i++) hipStreamDestroy(streams[i]);

    daemon_free();
    free(streams);

    return 0;    
}







