#include "matrixMul_kernel.cu"
#include <stdio.h>

//headers

//main
int main(){
  hipStream_t s;

  hipStreamCreate(&s);

  char *fileIn = "Inputs/matrixIn0.txt";;

  void* setupResults = matMul_setup(s, fileIn);
  matrixMul(s, setupResults);
  matMul_finish(s, fileIn, setupResults);

  //sync stream
  hipStreamSynchronize(s);

  // destroy stream
  hipStreamDestroy(s);
  
  return 0;
}
