#include "matrixMul_kernel.cu"
#include <stdio.h>

//headers

//main
int main(){
  
  int jobs = 25600;
  int throttle = 1;

  // create streams as many throttle
  hipStream_t* c = (hipStream_t* )malloc(throttle*sizeof(hipStream_t));
  int i;
  for(i=0;i<throttle;i++){
    hipStreamCreate(&c[i]);
  }

  // set input file
  char *fileIn = "Inputs/matrixIn0.txt";;

  // do work for each job
  int k;
  void* setupResults = matMul_setup(c[0], fileIn);
  hipDeviceSynchronize();

  for(k=0;k<jobs;k++){
    matrixMul(c[k%throttle], setupResults);
  }
  matMul_finish(c[0], fileIn, setupResults);  

  // sync device
  hipDeviceSynchronize();

  // destroy streams
  int j;
  for(j=0;j<throttle;j++){
    hipStreamDestroy(c[j]);
  }
  
  // free array
  free(c);

  // return
  return 0;
}
