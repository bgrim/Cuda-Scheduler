#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void clock_block(int kernel_time, int clockRate)
{ 
    int finish_clock;
    int start_time;
    for(int temp=0; temp<kernel_time; temp++){
        start_time = clock();
        finish_clock = start_time + clockRate;
        bool wrapped = finish_clock < start_time;
        while( clock() < finish_clock || wrapped) wrapped = clock()>0 && wrapped;
    }
}

void *sleep(void *str){
    hipStream_t stream = (hipStream_t) str;
    int kernel_time = 1000;

    int cuda_device = 0;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&cuda_device);	
    hipGetDeviceProperties(&deviceProp, cuda_device);
    int clockRate = deviceProp.clockRate;

    clock_block<<<1,1,1,stream>>>(kernel_time, clockRate);

    hipError_t cuda_error = hipStreamSynchronize(stream);
    if(cuda_error==hipSuccess){
        printf( "Running the Scheduler was a success\n");
    }else{
        printf("CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }
    return 0;
}
